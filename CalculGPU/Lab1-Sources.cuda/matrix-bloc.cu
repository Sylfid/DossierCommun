#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

#define TILE_WIDTH 16

// Charge une matrice disponible dans les repertoires exemples
bool load_matrix(char * filename, double * &matrix, int &nx, int &ny){
  std::string line;
  std::ifstream infile(filename);

  if (!infile.is_open()) {
    std::cout << "Fichier introuvable: "<< filename << std::endl;
    return 0;
  }

  // Charge la taile de la matrice
  infile >> nx >> ny;

  // Alloue le tableau correspondant
  matrix = new double[nx*ny];

  // Charge la matrice
  for (int i=0; i< nx*ny; i++){
    infile >> matrix[i];
  }

  infile.close();

  return 1;
}

// Calcul C = A * B
__global__ void matrixMultiply(float * A, float * B, float * C,
             int numARows, int numAColumns,
             int numBRows, int numBColumns,
             int numCRows, int numCColumns) {
  const int TILE_WIDTH = 32;
  __shared__ float sharedA[TILE_WIDTH][TILE_WIDTH];
  __shared__ float sharedB[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int Row = by*TILE_WIDTH + ty;
  int Col = bx*TILE_WIDTH + tx;
  float Cvalue = 0.0;
  if (numAColumns != numBRows) return ;
  for (int i = 0; i < (int)(ceil((float)numAColumns/TILE_WIDTH)); i++)
  {

    if (i*TILE_WIDTH + tx < numAColumns && Row < numARows){
      sharedA[ty][tx] = A[Row*numAColumns + i*TILE_WIDTH + tx];
    }else{
      sharedA[ty][tx] = 0.0;
    }

    if (i*TILE_WIDTH + ty < numBRows && Col < numBColumns){
      sharedB[ty][tx] = B[(i*TILE_WIDTH + ty)*numBColumns + Col];
    }else{
      sharedB[ty][tx] = 0.0;
    }
    __syncthreads();
    if(Row < numARows && Col < numBColumns){

      for(int j = 0; j < TILE_WIDTH; j++)
        Cvalue += sharedA[ty][j] * sharedB[j][tx];
    }

    __syncthreads();
  }

  if (Row < numCRows && Col < numCColumns)
    C[Row*numCColumns + Col] = Cvalue;
}

int main(int argc, char ** argv) {
    float * hostA;
    float * hostB;
    float * hostC;
    float * deviceA;
    float * deviceB;
    float * deviceC;
    int numARows;
    int numAColumns;
    int numBRows;
    int numBColumns;
    int numCRows;
    int numCColumns;


    hostA = (float *) wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
    hostB = (float *) wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
    /// Charger le fichier d'entree
    /// Initialiser numCRows et numCColumns
    numCRows = 0;
    numCColumns = 0;
    numCRows = numARows;
    numCColumns = numBColumns;
    /// Allouer hostC
    hostC = (float*) malloc(sizeof(float)*numCRows*numCColumns);

    /// Allouer la memoire sur GPU
    hipMalloc((void**)&deviceA , sizeof(float)*numARows*numAColumns );
    hipMalloc((void**)&deviceB , sizeof(float)*numBRows*numBColumns);
    hipMalloc((void**)&deviceC , sizeof(float)*numCRows*numCColumns);

    /// Copier la memoire sur le GPU
    hipMemcpy(deviceA, hostA, sizeof(float)*numARows*numAColumns, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, sizeof(float)*numBRows*numBColumns, hipMemcpyHostToDevice);

    /// Initialise la grille et les dimensions de chaque bloc
    int dimX = (int)(ceil((float)numCColumns / TILE_WIDTH));
    int dimY = (int)(ceil((float)numCRows / TILE_WIDTH));
    dim3 DimGrid(dimX, dimY);
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH);

    /// Execute le kernel
    matrixMultiplyShared<<<DimGrid , DimBlock>>>(deviceA , deviceB , deviceC , numARows , numAColumns, numBRows ,numBColumns , numCRows , numCColumns);

    hipDeviceSynchronize();

    /// Charge le resultat en memoire CPU
    hipMemcpy(hostC, deviceC, sizeof(float)*numCRows*numCColumns , hipMemcpyDeviceToHost);

    /// Libere la memoire
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    free(hostA);
    free(hostB);
    free(hostC);

   :1
     return 0;
}

