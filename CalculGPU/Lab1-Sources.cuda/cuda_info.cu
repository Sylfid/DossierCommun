#include  <hip/hip_runtime.h>

int main(int argc, char ** argv) {
    int deviceCount;

    hipGetDeviceCount(&deviceCount);

    for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;

        hipGetDeviceProperties(&deviceProp, dev);

        if (dev == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
               // Ne detecte pas CUDA
                return -1;
            } else {
              // Afficher le nombre de device
            }
        }

        // Afficher le nom de la device
        // Donner le numero de version majeur et mineur
        // Donner la taille de la memoire globale
        // Donner la taille de la memoire constante
        // Donner la taille de la memoire partagee par bloc
        // Donner le nombre de thread max dans chacune des directions
        // Donner le taille maximum de la grille pour chaque direction
        // Donner la taille du warp
    }

    return 0;
}
