#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

// Charge une matrice disponible dans les repertoires exemples
bool load_matrix(char * filename, double * &matrix, int &nx, int &ny){
  std::string line;
  std::ifstream infile(filename);

  if (!infile.is_open()) {
    std::cout << "Fichier introuvable: "<< filename << std::endl;
    return 0;
  }

  // Charge la taile de la matrice
  infile >> nx >> ny;

  // Alloue le tableau correspondant
  matrix = new double[nx*ny];

  // Charge la matrice
  for (int i=0; i< nx*ny; i++){
    infile >> matrix[i];
  }

  infile.close();

  return 1;
}

// Calcul C = A * B
__global__ void matrixMultiply(float * A, float * B, float * C,
             int numARows, int numAColumns,
             int numBRows, int numBColumns,
             int numCRows, int numCColumns) {
    /// Insérer le code
}

int main(int argc, char ** argv) {
    float * hostA;
    float * hostB;
    float * hostC;
    float * deviceA;
    float * deviceB;
    float * deviceC;
    int numARows;
    int numAColumns;
    int numBRows;
    int numBColumns;
    int numCRows;
    int numCColumns;



    /// Charger le fichier d'entree
    /// Initialiser numCRows et numCColumns
    numCRows = 0;
    numCColumns = 0;
    /// Allouer hostC

    /// Afficher les informations sur la matrice
    /// Allouer la memoire sur GPU

    /// Copier la memoire sur le GPU


    /// Initialise la grille et les dimensions de chaque bloc

    /// Execute le kernel

    hipDeviceSynchronize();

    /// Charge le resultat en memoire CPU

    /// Libere la memoire

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}

